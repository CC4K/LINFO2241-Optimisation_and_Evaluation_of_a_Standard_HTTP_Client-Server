#include "hip/hip_runtime.h"
/*
    This file is compiled only if the flag SIMT is given
*/

/* By default, nvcc compiles with C++ linkage, but we want C linkage */
extern "C" {

#include <stdint.h>
#include <stdlib.h>
#include <stdio.h>

#include "simt.h"

#ifndef CUDA_BLOCK_SIZE
    #define CUDA_BLOCK_SIZE 8
#endif



/*
    GPU kernel performing a matrix multiplication
    __global__ specifies that it will execute on the device (= the GPU)
 */
//  CUDA MATRIX MULT: https://stackoverflow.com/questions/18815489/cuda-tiled-matrix-matrix-multiplication-with-shared-memory-and-matrix-size-whic
 __global__ void kernel_multiply_matrix(uint32_t *A, uint32_t *B, uint32_t *C, uint32_t K) {

    uint32_t CValue = 0;
    
    int Row = blockIdx.y*blockDim.y + threadIdx.y;
    int Col = blockIdx.x*blockDim.y + threadIdx.x;

    __shared__ uint32_t As[CUDA_BLOCK_SIZE][CUDA_BLOCK_SIZE];
    __shared__ uint32_t Bs[CUDA_BLOCK_SIZE][CUDA_BLOCK_SIZE];


    for (int k = 0; k < (CUDA_BLOCK_SIZE + K - 1)/CUDA_BLOCK_SIZE; k++) {

            if (k*CUDA_BLOCK_SIZE + threadIdx.x < K && Row < K)
                As[threadIdx.y][threadIdx.x] = A[Row*K + k*CUDA_BLOCK_SIZE + threadIdx.x];
            else
                As[threadIdx.y][threadIdx.x] = 0;

            if (k*CUDA_BLOCK_SIZE + threadIdx.y < K && Col < K)
                Bs[threadIdx.y][threadIdx.x] = B[(k*CUDA_BLOCK_SIZE + threadIdx.y)*K + Col];
            else
                Bs[threadIdx.y][threadIdx.x] = 0;

            __syncthreads(); //prevents race conditions for shared memory
            
            for (int n = 0; n < CUDA_BLOCK_SIZE; ++n)
            CValue += As[threadIdx.y][n] * Bs[n][threadIdx.x];
            
            __syncthreads();
    }
        
    if (Row < K && Col < K)
        C[((blockIdx.y * blockDim.y + threadIdx.y)*K) +
            (blockIdx.x * blockDim.x)+ threadIdx.x] = CValue;
            
}




__global__ void kernel_test_patterns(uint32_t *matrix, uint32_t m, uint32_t *patterns, uint32_t pattern_size, uint32_t nb_patterns, uint32_t *res) {

    uint32_t i = blockIdx.y * blockDim.y + threadIdx.y;
    uint32_t j = blockIdx.x * blockDim.x + threadIdx.x;
    
    
    if ( i < (m - pattern_size + 1) && j < nb_patterns) {
        uint32_t dist = 0;
        uint32_t new_j = j * pattern_size;

        uint32_t k;
        for (k = 0; k + 7 < pattern_size; k += 8) {
            dist += (matrix[i + k] - patterns[new_j + k]) * (matrix[i + k] - patterns[new_j + k]);
            dist += (matrix[i + k + 1] - patterns[new_j + k + 1]) * (matrix[i + k + 1] - patterns[new_j + k + 1]);
            dist += (matrix[i + k + 2] - patterns[new_j + k + 2]) * (matrix[i + k + 2] - patterns[new_j + k + 2]);
            dist += (matrix[i + k + 3] - patterns[new_j + k + 3]) * (matrix[i + k + 3] - patterns[new_j + k + 3]);
            dist += (matrix[i + k + 4] - patterns[new_j + k + 4]) * (matrix[i + k + 4] - patterns[new_j + k + 4]);
            dist += (matrix[i + k + 5] - patterns[new_j + k + 5]) * (matrix[i + k + 5] - patterns[new_j + k + 5]);
            dist += (matrix[i + k + 6] - patterns[new_j + k + 6]) * (matrix[i + k + 6] - patterns[new_j + k + 6]);
            dist += (matrix[i + k + 7] - patterns[new_j + k + 7]) * (matrix[i + k + 7] - patterns[new_j + k + 7]);
        }

        for (; k < pattern_size; k++) {
            dist += (matrix[i + k] - patterns[new_j + k]) * (matrix[i + k] - patterns[new_j + k]);
        }
        
        atomicMin(&res[j], dist);
    }
    
}



uint32_t *cuda_memory = nullptr;
size_t allocated_size = 0;

/*
    Helper function that allocates GPU memory, copies the data to the GPU, and launches the kernel 
    It will execute on the host (= the CPU)

    Two functions were merged into one to optimize performance
*/
void multiply_matrix_and_test_patterns_simt(uint32_t *matrix1, uint32_t *matrix2, uint32_t K, uint32_t *patterns, uint32_t pattern_size, uint32_t nb_patterns, uint32_t *res){
    uint32_t matrix_size_1D = K * K;
    
    uint32_t  *A_cuda, *B_cuda, *C_cuda, *patterns_cuda, *res_cuda;

    uint32_t size_to_allocate_matrix = 3 * matrix_size_1D;
    uint32_t size_to_allocate_patterns = matrix_size_1D + nb_patterns + nb_patterns * pattern_size;
    size_t required_size = max(size_to_allocate_matrix, size_to_allocate_patterns);

    // CudaMalloc is a big bottleneck so it's usage is minimized to the strict minimum
    if (required_size > allocated_size) {
        if (cuda_memory != nullptr) {
            hipFree(cuda_memory);
        }
        hipMalloc(&cuda_memory, required_size * sizeof(uint32_t));
        allocated_size = required_size;
    }

    A_cuda = cuda_memory + 2 * matrix_size_1D;
    B_cuda = cuda_memory + matrix_size_1D;
    C_cuda = cuda_memory;

    patterns_cuda = cuda_memory + matrix_size_1D;
    res_cuda = cuda_memory + matrix_size_1D + nb_patterns*pattern_size;

    /* Copy matrices A and B from host to device */
    hipMemcpy(A_cuda, matrix1, K * K * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(B_cuda, matrix2, K * K * sizeof(uint32_t), hipMemcpyHostToDevice);
    
    /* Define the block and grid dimensions */
    dim3 threadsPerBlock(CUDA_BLOCK_SIZE, CUDA_BLOCK_SIZE); 
    dim3 numBlocks((K + threadsPerBlock.x - 1) / threadsPerBlock.x, (K + threadsPerBlock.y - 1) / threadsPerBlock.y);  
    
    /* Launch the kernel */
    kernel_multiply_matrix <<< numBlocks, threadsPerBlock >>> (A_cuda, B_cuda, C_cuda, K);

    /* Wait for the kernel to finish */
    hipDeviceSynchronize();

    hipMemcpy(patterns_cuda, patterns, nb_patterns * pattern_size * sizeof(uint32_t), hipMemcpyHostToDevice);
    
    hipMemset(res_cuda, UINT32_MAX, nb_patterns * sizeof(uint32_t));
    
    dim3 numBlocks_patterns((nb_patterns + threadsPerBlock.x - 1) / threadsPerBlock.x, ((matrix_size_1D - pattern_size + 1) + threadsPerBlock.y - 1) / threadsPerBlock.y); 
    
    // the matrix from the multiplication while keeping it on the GPU
    kernel_test_patterns<<<numBlocks_patterns, threadsPerBlock>>>(C_cuda, matrix_size_1D, patterns_cuda, pattern_size, nb_patterns, res_cuda);

    hipMemcpy(res, res_cuda, nb_patterns * sizeof(uint32_t), hipMemcpyDeviceToHost);

}

} /* extern "C" */
